#include "hip/hip_runtime.h"
#include "./../headers/data_structures.h"
#include "./../headers/CUDA_functions.cuh"

__host__
void freeAll_GPU(){
	hipFree(POLY_COEFFS_GPU);
	hipFree(POLY_EXPONS_GPU);
	hipFree(iter_matrix_GPU);
	hipFree(roots_matrix_GPU);
	hipFree(cnumber_matrix_GPU);
	return;
}


__device__ double carg(const hipDoubleComplex& z) {
	return (double)atan2(hipCimag(z), hipCreal(z));
} 

__device__ hipDoubleComplex cpow(const hipDoubleComplex& z, const double& n) {
	return make_hipDoubleComplex((pow(hipCabs(z), n)*cos(n*carg(z))), (pow(hipCabs(z), n)*sin(n*carg(z))));
}

__device__ hipDoubleComplex clog_cu(const hipDoubleComplex& z) {
	return make_hipDoubleComplex( log(hipCabs(z)), carg(z) );
}

__device__ hipDoubleComplex cexp(const hipDoubleComplex& arg){
   hipDoubleComplex res;
   float s, c;
   float e = expf(arg.x);
   sincosf(arg.y, &s, &c);
   res.x = c * e;
   res.y = s * e;
   return res;
}


// Complex Gammafunction
__device__
hipDoubleComplex cgammln(double xx)
{
	hipDoubleComplex minusOne = make_hipDoubleComplex(-1.0f,0);
	hipDoubleComplex xxx,yyy,tmp,ser,res;
	static double cof[6]={76.18009172947146,-86.50532032941677,
		24.01409824083091,-1.231739572450155,
		0.1208650973866179e-2,-0.5395239384953e-5};
	int j;
	xxx.x=xx;
	xxx.y=0;
	yyy.x=xx;
	yyy.y=0; 
	tmp.x = xxx.x + 5.5;
	tmp.y = 0;
	tmp=hipCadd(tmp,hipCmul(hipCmul(hipCadd(xxx,make_hipDoubleComplex(0.5,0)),make_hipDoubleComplex(log(tmp.x),0)),minusOne));
	ser=make_hipDoubleComplex(1.000000000190015,0);
	for (j=0;j<=5;j++){
		yyy = hipCadd(yyy,make_hipDoubleComplex(1.0f,0));
		ser =hipCadd(ser, hipCdiv( make_hipDoubleComplex(cof[j],0),yyy));
	}
	res.x = -tmp.x+hipCreal(clog_cu(make_hipDoubleComplex(2.5066282746310005*ser.x/xxx.x,0)));
	res.y = 0.0;
	return res;
}


__device__
hipDoubleComplex get_fdf_GPU(hipDoubleComplex& z, double* params, int n_terms, double* exponents,int FRAC_DER_MODE, double ALPHA_FRAC_DER ){
	hipDoubleComplex minusOne = make_hipDoubleComplex(-1.0f,0);
	hipDoubleComplex f = make_hipDoubleComplex(0,0);
	hipDoubleComplex df = make_hipDoubleComplex(0,0);
	hipDoubleComplex res = make_hipDoubleComplex(0,0);
	for(int i = n_terms-1; i>-1;i--){
		hipDoubleComplex rationalExponent = make_hipDoubleComplex(exponents[i],0);
		hipDoubleComplex temp1 = cpow(z,hipCreal(rationalExponent));
		f = hipCadd(f, hipCmul(make_hipDoubleComplex(params[i],0),temp1));		
	}
	// Normal derivative
	if(FRAC_DER_MODE==0){
		for(int i = n_terms-1; i>-1;i--){
			hipDoubleComplex rationalExponent = make_hipDoubleComplex(exponents[i],0);
			if(abs(params[i]) > 0.00000000001){
				hipDoubleComplex temp2 = cpow(z,hipCreal(hipCadd(rationalExponent,minusOne)));
				df = hipCadd(df,hipCmul(hipCmul(make_hipDoubleComplex(params[i],0),rationalExponent),temp2));
			}
		}
	}
	// Riemann-Liouville derivative
	else{
		for(int i = n_terms-1; i>-1;i--){
			if((abs(params[i]) > 0.00000000001)){
				if( (abs(exponents[i])  > 0.00000000001)) {
					hipDoubleComplex temp2 = cpow(z,abs(int(exponents[i]))-ALPHA_FRAC_DER);
					hipDoubleComplex temp3 = cgammln(abs(int(exponents[i]))-ALPHA_FRAC_DER+1.0);
					hipDoubleComplex temp4 = cgammln(abs(int(exponents[i]))+1.0);
					df = hipCadd(df, hipCmul(make_hipDoubleComplex( hipCreal( hipCdiv(cexp(temp4),cexp(temp3))),hipCimag(hipCdiv(cexp(temp4),cexp(temp3)))),temp2));
				}
				else{
					hipDoubleComplex temp2 = cpow(z,-ALPHA_FRAC_DER);
					hipDoubleComplex temp3 = cgammln(1.0-ALPHA_FRAC_DER);
					df = hipCadd(df,hipCdiv(temp2, make_hipDoubleComplex(hipCreal(cexp(temp3)),hipCimag(cexp(temp3)))));
				}
			}
		}
	}
	res = hipCmul(minusOne,hipCdiv(f,df));
	return res;
}

__global__
void solve_cnewton_iter_GPU(int* iter_matrix_GPU, hipDoubleComplex* cnumber_matrix_GPU, double accuracy, int max_iterations, double * params, int n_terms, double * rationalExponents,int N_X,int N_Y,int FRAC_DER_MODE, double ALPHA_FRAC_DER ){
	
	// printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int threadStride = blockDim.x * gridDim.x;
	hipDoubleComplex fdf = make_hipDoubleComplex(0,0);
	int matInd =threadIndex; 
	hipDoubleComplex z = make_hipDoubleComplex(hipCreal(cnumber_matrix_GPU[matInd]),hipCimag(cnumber_matrix_GPU[matInd]));
	for(int i = 0; i < max_iterations; i++){
		fdf = get_fdf_GPU(z,params,n_terms,rationalExponents,FRAC_DER_MODE,ALPHA_FRAC_DER);
		z = hipCadd(z,fdf);
		if(hipCabs(fdf) < accuracy){
			iter_matrix_GPU[matInd] = i;
			break;
		}
		else{
			iter_matrix_GPU[matInd] = max_iterations;
		}
	}
	return;
}

__global__
void solve_cnewton_roots_GPU(hipDoubleComplex* roots_matrix_GPU, hipDoubleComplex* cnumber_matrix_GPU, double accuracy, int max_iterations, double * params, int n_terms, double * rationalExponents,int N_X,int N_Y,int FRAC_DER_MODE, double ALPHA_FRAC_DER ){
	// printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int threadStride = blockDim.x * gridDim.x;
	hipDoubleComplex fdf = make_hipDoubleComplex(0,0);
	int matInd =threadIndex; 
	hipDoubleComplex z = make_hipDoubleComplex(hipCreal(cnumber_matrix_GPU[matInd]),hipCimag(cnumber_matrix_GPU[matInd]));
	for(int i = 0; i < max_iterations; i++){
		fdf = get_fdf_GPU(z,params,n_terms,rationalExponents,FRAC_DER_MODE,ALPHA_FRAC_DER );
		z = hipCadd(z,fdf);
		if(hipCabs(fdf) < accuracy){
			roots_matrix_GPU[matInd] = z;
			break;
		}
		else{
			roots_matrix_GPU[matInd] = make_hipDoubleComplex(0,0);
		}
	}
	return;
}


__host__
void setup_GPU(){
	numBlocks = (N_X*N_Y + BLOCK_SIZE - 1) / BLOCK_SIZE;
	POLY_COEFFS_GPU = new double[POLY_N_TERMS]();
	POLY_EXPONS_GPU = new double[POLY_N_TERMS]();
	cnumber_matrix_GPU = new hipDoubleComplex[N_X*N_Y];
	iter_matrix_GPU = new int[N_X*N_Y];
	roots_matrix_GPU = new hipDoubleComplex[N_X*N_Y];
	hipMallocManaged(&POLY_COEFFS_GPU, (POLY_N_TERMS)*sizeof(double));
	hipMallocManaged(&POLY_EXPONS_GPU, (POLY_N_TERMS)*sizeof(double));
	hipMallocManaged(&iter_matrix_GPU, (N_X*N_Y)*sizeof(int));
	hipMallocManaged(&cnumber_matrix_GPU, (N_X*N_Y)*sizeof(hipDoubleComplex));
	hipMallocManaged(&roots_matrix_GPU, (N_X*N_Y)*sizeof(hipDoubleComplex));
	for (int i = 0; i < POLY_N_TERMS; i++){
		POLY_COEFFS_GPU[i] = POLY_COEFFS[i];
		POLY_EXPONS_GPU[i] = POLY_EXPONS[i];
	}
	return;
}

__host__
void fillCnumberMatrixGPU(){
	for (int i = 0; i<N_Y; i++){
		for (int j = 0; j<N_X; j++){
			cnumber_matrix_GPU[i*N_X+j] = make_hipDoubleComplex(real(cnumber_matrix[i][j]),(imag(cnumber_matrix[i][j])));
		}
	}
	return;
}

__host__
void getIterMatrixGPU(vector<vector<float>>& iterImage){
	solve_cnewton_iter_GPU<<<numBlocks,BLOCK_SIZE>>>(iter_matrix_GPU,cnumber_matrix_GPU,ACCURACY,N_ITER,POLY_COEFFS_GPU,POLY_N_TERMS,POLY_EXPONS_GPU,N_X,N_Y,FRAC_DER_MODE,ALPHA_FRAC_DER);
	hipDeviceSynchronize();
	for(int i=0; i<N_Y; i++){
		vector<float> row;
		for(int j=0; j<N_X; j++){
			row.push_back(log(iter_matrix_GPU[j+i*N_X]));
		}
		iterImage.push_back(row);
	}
	return;
}

__host__
void getRootsMatrixGPU(vector<vector<float>>& rootsImage){
	solve_cnewton_roots_GPU<<<numBlocks,BLOCK_SIZE>>>(roots_matrix_GPU,cnumber_matrix_GPU,ACCURACY,N_ITER,POLY_COEFFS_GPU,POLY_N_TERMS,POLY_EXPONS_GPU,N_X,N_Y,FRAC_DER_MODE,ALPHA_FRAC_DER);
	hipDeviceSynchronize();
	vector<complex<double>> roots;

	for(int i=0; i<N_Y; i++){
		vector<float> row;
		for(int j=0; j<N_X; j++){
			complex<double> root = {hipCreal(roots_matrix_GPU[j+i*N_X]),hipCimag(roots_matrix_GPU[j+i*N_X])};
			if(abs(real(root)) > 10*INTERVAL_LENGTH || abs(imag(root)) > 10*INTERVAL_LENGTH ){
				root = {0,0};
			} 
			if(i==0 && j==0){
				roots.push_back(root);
			}
			bool new_root = true;
			for (int k=0; k< roots.size(); k++){
				if( abs(real(root)-real(roots[k]))<0.01 && abs(imag(root)-imag(roots[k]))<0.01){
					roots_matrix[i][j]=k;
					row.push_back(roots_matrix[i][j]);
					new_root=false;
					break;
				}
			}
			if(new_root==true){
				roots_matrix[i][j]=roots.size()+1;
				row.push_back(roots_matrix[i][j]);
				roots.push_back(root);
			}
		}
		rootsImage.push_back(row);
	}
	cout << "Found " << roots.size() << " roots:" << endl;
	for (int k=0; k< roots.size(); k++){
		cout << " " << real(roots[k]) << " " << imag(roots[k]) << endl;
	}
	return;
}